#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <math.h>

#include "image_reconstract_GPU.h"


#include "kernel.h"
 #define TX 32
#define TY 32

//////////////
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<iostream>

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>


using namespace std;
typedef unsigned char BYTE;
/*struct PixelDefine
{
	long offset;
	unsigned short line_index;
	unsigned short sample_index;
	float c1, c2, c3, c4;
};*/

//BW
#define FILTER_LENGTH 3
#define no_of_threads 256
#define no_of_blocks 1
#define FILESIZE 2992128
#define N_SMPL 11688
#define FIR_FILTER_LENGTH 9
#define FIR_FILTER_LENGTH_D2 (FIR_FILTER_LENGTH/2)
#define COL_PIXEL 512
/*#define  R  61
#define   pitch  0.2
#define  depth  180
#define  lens  0.7
#define  nLines  256
#define  nSamples  512*/

#define counter 173621
#define itration_thread 43 // 43//172 //347
#define no_of_threads_interpolation 1009//500
#define no_of_blocks_interpolation  4

#define CHANNELCOUNT 3
#define TOTAL_SIZE 11968512
#define TOTAL_PIXEL 512*512
#define IMSIZE 131072



__device__ unsigned char clip(int n)
{ return n > 255 ? 255 : (n < 0 ? 0 : n); }

 __global__ void  distanceKernel(uchar4 *d_out, int w, int h, int2 pos) {
	  const int c = blockIdx.x*blockDim.x + threadIdx.x;
  const int r = blockIdx.y*blockDim.y + threadIdx.y;
	   if ((c >= w) || (r >= h)) return; // Check if within image bounds
	   const int i = c + r*w; // 1D indexing
	   const int dist = sqrtf((c - pos.x)*(c - pos.x) + (r - pos.y)*(r - pos.y));
   const unsigned char intensity = clip(255 - dist);
	   d_out[i].x = intensity;
	 d_out[i].y = intensity;
	d_out[i].z = 0;
	  d_out[i].w = 255;

}

void kernelLauncher(uchar4 *d_out, int w, int h, int2 pos) {
	 const dim3 blockSize(TX, TY);
   const dim3 gridSize = dim3((w + TX - 1) / TX, (h + TY - 1) / TY);
  distanceKernel << <gridSize, blockSize >> >(d_out, w, h, pos);

}
//////////////////////



//low
__global__ void bw_lowpass_filterKernel(float *in, float *output, float *output2)
{
	float b1[FILTER_LENGTH];
	float a1[FILTER_LENGTH];
	b1[0] = 1.0000;
	b1[1] = 2.0000;
	b1[2] = 1.0000;
	a1[0] = 1.0000;
	a1[1] = -1.2246515810130951;
	a1[2] = 0.45044543005604082;
	float g1 = 0.056448462260736451;
	float d1[3] = { 0,0,0 };

	int j = threadIdx.y;


	for (int i = 0; i <N_SMPL; i++)

	{
		d1[2] = d1[1];
		d1[1] = d1[0];
		d1[0] = in[i + j*N_SMPL] - a1[1] * d1[1] - a1[2] * d1[2];
		output[i + j*N_SMPL] = g1*b1[0] * d1[0] + g1*b1[1] * d1[1] + g1*b1[2] * d1[2];


	}
	float b[FILTER_LENGTH];
	float a[FILTER_LENGTH];
	b[0] = 1.0000;
	b[1] = -2.0000;
	b[2] = 1.0000;
	a[0] = 1.0000;
	a[1] = -1.5610180758007182;
	a[2] = 0.64135153805756306;
	float g = 0.75502196198037386;
	float d[3] = { 0,0,0 };


	//for ( j = 0; j <  no_of_threads; j++)
	//{
	for (int i = 0; i <N_SMPL; i++)
		//d1[3] = d1[2];
	{
		d[2] = d[1];
		d[1] = d[0];
		d[0] = output[i + j*N_SMPL] - a[1] * d[1] - a[2] * d[2];
		output2[i + j*N_SMPL] = g*b[0] * d[0] + g*b[1] * d[1] + g*b[2] * d[2];
	}

}

__global__ void bw_highpass_filterKernel(float *in, float *output, float *output2)
{
	float b[FILTER_LENGTH];
	float a[FILTER_LENGTH];
	b[0] = 1.0000;
	b[1] = -2.0000;
	b[2] = 1.0000;
	a[0] = 1.0000;
	a[1] = -1.5610180758007182;
	a[2] = 0.64135153805756306;
	float g = 0.75502196198037386;
	float d[3] = { 0,0,0 };
	int j = threadIdx.y;

	//for ( j = 0; j <  no_of_threads; j++)
	//{
	for (int i = 0; i <N_SMPL; i++)
		//d1[3] = d1[2];
	{
		d[2] = d[1];
		d[1] = d[0];
		d[0] = in[i + j*N_SMPL] - a[1] * d[1] - a[2] * d[2];
		output[i + j*N_SMPL] = g*b[0] * d[0] + g*b[1] * d[1] + g*b[2] * d[2];
	}
	//}

	float b1[FILTER_LENGTH];
	float a1[FILTER_LENGTH];
	b1[0] = 1.0000;
	b1[1] = 2.0000;
	b1[2] = 1.0000;
	a1[0] = 1.0000;
	a1[1] = -1.2246515810130951;
	a1[2] = 0.45044543005604082;
	float g1 = 0.056448462260736451;
	float d1[3] = { 0,0,0 };

	//	int j = threadIdx.y;


	for (int i = 0; i <N_SMPL; i++)

	{
		d1[2] = d1[1];
		d1[1] = d1[0];
		d1[0] = output[i + j*N_SMPL] - a1[1] * d1[1] - a1[2] * d1[2];
		output2[i + j*N_SMPL] = g1*b1[0] * d1[0] + g1*b1[1] * d1[1] + g1*b1[2] * d1[2];


	}


}

__global__ void ellaptic_lowpass_filterKernel(float *in, float *output)
{
	float b1[FILTER_LENGTH];
	float a1[FILTER_LENGTH];
	b1[0] = 1.0000;
	b1[1] = 1.9951831916798526;
	b1[2] = 1.0000;
	a1[0] = 1.0000;
	a1[1] = -1.2846437324109372;
	a1[2] = 0.54828472668060735;
	float	g1 = 0.058813394079824187;
	float d1[3] = { 0,0,0 };

	int j = threadIdx.y;


	for (int i = 0; i <N_SMPL; i++)

	{
		d1[2] = d1[1];
		d1[1] = d1[0];
		d1[0] = in[i + j*N_SMPL] - a1[1] * d1[1] - a1[2] * d1[2];
		output[i + j*N_SMPL] = g1*b1[0] * d1[0] + g1*b1[1] * d1[1] + g1*b1[2] * d1[2];


	}


}

__global__ void ellaptic_highpass_filterKernel(float *in, float *output)
{
	float b[FILTER_LENGTH];
	float a[FILTER_LENGTH];
	b[0] = 1.0000;
	b[1] = -1.9999897887443692;
	b[2] = 1.0000;
	a[0] = 1.0000;
	a[1] = -1.2846437324109372;
	a[2] = 0.73284143556340198;
	float	g = 0.80059240346457028;

	float d[3] = { 0,0,0 };
	int j = threadIdx.y;

	//for ( j = 0; j <  no_of_threads; j++)
	//{
	for (int i = 0; i <N_SMPL; i++)
		//d1[3] = d1[2];
	{
		d[2] = d[1];
		d[1] = d[0];
		d[0] = in[i + j*N_SMPL] - a[1] * d[1] - a[2] * d[2];
		output[i + j*N_SMPL] = g*b[0] * d[0] + g*b[1] * d[1] + g*b[2] * d[2];
	}
	//}


}
__global__ void chebychev_lowpass_filterKernel(float *in, float *output)
{
	float b1[FILTER_LENGTH];
	float a1[FILTER_LENGTH];
	b1[0] = 1.0000;
	b1[1] = 2;
	b1[2] = 1.0000;
	a1[0] = 1.0000;
	a1[1] = -1.2846384667454742;
	a1[2] = 0.5482635908391793;
	float g1 = 0.065906281023426286;

	float d1[3] = { 0,0,0 };

	int j = threadIdx.y;


	for (int i = 0; i <N_SMPL; i++)

	{
		d1[2] = d1[1];
		d1[1] = d1[0];
		d1[0] = in[i + j*N_SMPL] - a1[1] * d1[1] - a1[2] * d1[2];
		output[i + j*N_SMPL] = g1*b1[0] * d1[0] + g1*b1[1] * d1[1] + g1*b1[2] * d1[2];


	}


}

__global__ void chebychev_highpass_filterKernel(float *in, float *output)
{
	float b[FILTER_LENGTH];
	float a[FILTER_LENGTH];
	b[0] = 1.0000;
	b[1] = -2;
	b[2] = 1.0000;
	a[0] = 1.0000;
	a[1] = -1.6557169643711138;
	a[2] = 0.73281693212820698;
	float g = 0.8471334741248302;

	float d[3] = { 0,0,0 };
	int j = threadIdx.y;

	//for ( j = 0; j <  no_of_threads; j++)
	//{
	for (int i = 0; i <N_SMPL; i++)
		//d1[3] = d1[2];
	{
		d[2] = d[1];
		d[1] = d[0];
		d[0] = in[i + j*N_SMPL] - a[1] * d[1] - a[2] * d[2];
		output[i + j*N_SMPL] = g*b[0] * d[0] + g*b[1] * d[1] + g*b[2] * d[2];
	}
	//}


}
__global__ void least_lowpass_filterKernel(float *in, float *output)
{
	float b1[FILTER_LENGTH];
	float a1[FILTER_LENGTH];
	b1[0] = 1.0000;
	b1[1] = -1.5559137587684049;
	b1[2] = 0.99955810783813837;
	a1[0] = 1.0000;
	a1[1] = -1.6767078264281887;
	a1[2] = 0.860558332275835;
	float g1 = 0.29202831921765487;

	float d1[3] = { 0,0,0 };

	int j = threadIdx.y;


	for (int i = 0; i <N_SMPL; i++)

	{
		d1[2] = d1[1];
		d1[1] = d1[0];
		d1[0] = in[i + j*N_SMPL] - a1[1] * d1[1] - a1[2] * d1[2];
		output[i + j*N_SMPL] = g1*b1[0] * d1[0] + g1*b1[1] * d1[1] + g1*b1[2] * d1[2];
	}


}

__global__ void least_highpass_filterKernel(float *in, float *output)
{
	float b[FILTER_LENGTH];
	float a[FILTER_LENGTH];
	b[0] = 1.0000;
	b[1] = -1.9605508523713233;
	b[2] = 0.99981455006498965;
	a[0] = 1.0000;
	a[1] = -1.7323249430855401;
	a[2] = 0.84774325464456513;
	float g = 0.69372279754577948;

	float d[3] = { 0,0,0 };
	int j = threadIdx.y;

	//for ( j = 0; j <  no_of_threads; j++)
	//{
	for (int i = 0; i <N_SMPL; i++)
		//d1[3] = d1[2];
	{
		d[2] = d[1];
		d[1] = d[0];
		d[0] = in[i + j*N_SMPL] - a[1] * d[1] - a[2] * d[2];
		output[i + j*N_SMPL] = g*b[0] * d[0] + g*b[1] * d[1] + g*b[2] * d[2];
	}
	//}


}

__global__ void constra_lowpass_filterKernel(float *in, float *output)
{
	float b1[FILTER_LENGTH];
	float a1[FILTER_LENGTH];
	b1[0] = 1.0000;
	b1[1] = -1.5393927524883004;
	b1[2] = 0.9999998000000101;
	a1[0] = 1.0000;
	a1[1] = -1.6644794016875226;
	a1[2] = 0.84640000000000004;
	float g1 = 0.28301493727636329;



	float d1[3] = { 0,0,0 };

	int j = threadIdx.y;


	for (int i = 0; i <N_SMPL; i++)

	{
		d1[2] = d1[1];
		d1[1] = d1[0];
		d1[0] = in[i + j*N_SMPL] - a1[1] * d1[1] - a1[2] * d1[2];
		output[i + j*N_SMPL] = g1*b1[0] * d1[0] + g1*b1[1] * d1[1] + g1*b1[2] * d1[2];
	}


}

__global__ void constra_highpass_filterKernel(float *in, float *output)
{
	float b[FILTER_LENGTH];
	float a[FILTER_LENGTH];
	b[0] = 1.0000;
	b[1] = -1.9609155894527821;
	b[2] = 0.9999998000000101;
	a[0] = 1.0000;
	a[1] = -1.7309485194818997;
	a[2] = 0.84640000000000004;
	float g = 0.69613172549115021;


	float d[3] = { 0,0,0 };
	int j = threadIdx.y;

	//for ( j = 0; j <  no_of_threads; j++)
	//{
	for (int i = 0; i <N_SMPL; i++)
		//d1[3] = d1[2];
	{
		d[2] = d[1];
		d[1] = d[0];
		d[0] = in[i + j*N_SMPL] - a[1] * d[1] - a[2] * d[2];
		output[i + j*N_SMPL] = g*b[0] * d[0] + g*b[1] * d[1] + g*b[2] * d[2];
	}
	//}


}
__global__ void detection_Kernel(float *final_output, float *x_out, float* z, float *final_output_PhasShift, float *det_out)
{

	static float b1[FIR_FILTER_LENGTH] = { 0 , -0.3706 , 0 , -0.6386 , 0 , 0.6386 , 0 , 0.3706 , 0 };
	*final_output_PhasShift = *final_output + FIR_FILTER_LENGTH_D2;
	int j = threadIdx.y;


	for (int i = 0; i < N_SMPL; i++)
		if (i < N_SMPL - 9)
		{
			{


				z = final_output + i + j* N_SMPL;
				float acc = 0;
				{
					acc = acc + b1[7] * z[0];
				}
				{
					acc = acc + b1[5] * z[2];
				}
				{
					acc = acc + b1[3] * z[4];
				}
				{
					acc = acc + b1[1] * z[6];
				}
				float q = final_output_PhasShift[i + j* N_SMPL];
				x_out[i + j* N_SMPL] = sqrt((acc*acc) + q*q);

			}
		}

	float g2 = 0.0036216815149286421;
	float b0 = 1, b3 = 2, b2 = 1;
	float a0 = 1, a1 = -1.8226949251963083, a2 = 0.83718165125602262;
	float a = 0, b = 0, c = 0, d;


	for (int i = 0; i <N_SMPL; i++)
		if (i < N_SMPL - 9)
		{
			{
				d = c;
				c = b;
				b = a;
				a = x_out[i + j*N_SMPL] - a1*b - a2*c;

				det_out[i + j*N_SMPL] = g2*b0*a + g2*b3*b + g2*b2*c;

			}
		}

}

__global__ void lpf_Kernel(float *x_out, float *det_out)
{
	int j = threadIdx.y;
	float g2 = 0.0036216815149286421;
	register float b0 = 1, b3 = 2, b2 = 1;
	register float a0 = 1, a1 = -1.8226949251963083, a2 = 0.83718165125602262;
	register float a = 0, b = 0, c = 0, d;


	for (int i = 0; i <N_SMPL; i++)
		//if (i < N_SMPL - 9)
		//{
	{
		d = c;
		c = b;
		b = a;
		a = x_out[i + j*N_SMPL] - a1*b - a2*c;

		det_out[i + j*N_SMPL] = g2*b0*a + g2*b3*b + g2*b2*c;

		//}
	}
}
__global__ void av_sampling_Kernel(float *det_out, float *samp_out)
{
	//int x = threadIdx.y;
	int i = threadIdx.y;
	int w = (i)*COL_PIXEL;
	float L = 0;
	float sum = 0;
	int j = 1;
	float 	average = 0;
	float window_size = (float)N_SMPL / (float)COL_PIXEL;
	for (int k = 0; k < N_SMPL; k++)
	{
		float NewL = floor((k / window_size) + 1);
		float dto = det_out[i*N_SMPL + k];
		sum = sum + dto;
		if (L == NewL)
		{
			j = j + 1;
			continue;
		}
		else
		{
			average = sum / j;
			samp_out[w] = average;
			//	samp_out[w + i*N_SMPL]= 255;
			w += 1;
			j = 0;
			sum = 0;
		}
		L = NewL;
	}
}
__global__ void peak_sampling_Kernel(float *det_out, float *psamp_out)
{
	/*int nt = omp_get_max_threads();
	//omp_set_num_threads(nt - 0);
	int undertype;
	/*if (undertype == 0)		                          //PEAK DETECT
	{*/
	float window_size = (float)N_SMPL / (float)COL_PIXEL;

	int i = threadIdx.y;
	float max = 0;
	float L = 0;
	int w = (i)*COL_PIXEL;
	for (int k = 0; k < N_SMPL; k++)
	{
		float NewL = floor((k / window_size) + 1);
		float dto = det_out[i*N_SMPL + k];
		if (max < dto)	max = dto;
		if (L != NewL)
		{
			psamp_out[w] = max;
			w += 1;
			max = 0;
		}
		L = NewL;
	}
}
__global__ void sampling_Kernel(float *det_out, float *ssamp_out)    //Sample
{

	float window_size = (float)N_SMPL / (float)COL_PIXEL;

	int i = threadIdx.y;
	float L = 0;
	int w = i*COL_PIXEL;
	for (int k = 0; k < N_SMPL; k++)
	{
		float NewL = floor((k / window_size) + 1);
		float dto = det_out[i*N_SMPL + k];
		if (L != NewL)
		{
			ssamp_out[w] = dto;
			w += 1;
		}
		L = NewL;
	}
}


__global__ void dynamic_Kernel(float *newscale, float *under_sampling_out, int *dB, float *dynamic_out, float *maxthreads)
{
	int dBrange[1] = { 0 };
	*dBrange = *dB;

	int dBmax = 150;
	float thresh = pow((float)10, (float)(dBmax - dBrange[0]) / 20);
	float ymax = 0;
	float maximum_y = 0;
	float max = 0;

	int j = threadIdx.y;
	/*	if (dBrange >= 150 || dBrange <= 0)
	{
	//this command could works in consol app only cout<<"ERROR dBrange must be between 0 and 150"<<endl;
	AfxMessageBox(_T("ERROR dBrange must be between 0 and 150"));
	dBrange = 60;
	}*/
	for (int i = 0; i < 512; i++)
	{
		dynamic_out[i + j * 512] = abs(under_sampling_out[i + j * 512]);

		/*if (maximum_y < dynamic_out[i])
		{
		maximum_y = dynamic_out[i];
		}*/

		if (maxthreads[j] < dynamic_out[i + j * 512])
		{
			maxthreads[j] = dynamic_out[i + j * 512];
		}
	}

	for (int i = 0; i<256; i++)
	{
		if (maximum_y < maxthreads[i])
		{
			maximum_y = maxthreads[i];
		}
	}

	float x = pow((float)10, (float) 7.5);
	ymax = maximum_y / x;

	int min1 = dBmax - dBrange[0];

	int newmax = 255;
	int newmin = 0;
	int oldrange = dBmax - min1;
	int newrange = newmax - newmin;

	for (int i = 0; i < 512; i++)
	{

		dynamic_out[i + j * 512] = dynamic_out[i + j * 512] / ymax;
		if (dynamic_out[i + j * 512] > thresh)
			dynamic_out[i + j * 512] = dynamic_out[i + j * 512];
		else
			dynamic_out[i + j * 512] = thresh;

		//	newscale[i + j * 512] = dynamic_out[i + j * 512];
		dynamic_out[i + j * 512] = 20 * log10(dynamic_out[i + j * 512]);
		newscale[i + j * 512] = (((dynamic_out[i + j * 512] - min1) * newrange) / oldrange) + newmin;
	}
}

// try
__global__ void interpolation_kernal(PixelDefine *ptrpixel_x, float *newscale, PixelDefine *s, BYTE *final_image)
{
	int	i = threadIdx.y;
	int b = blockIdx.y;

	//for (int m = 0; m <no_of_blocks_interpolation;m++)
	//{
	__shared__ PixelDefine ptrpixel[no_of_threads_interpolation];

	for (int j = 0; j < itration_thread; j++)
		//for (int j = 0; j <1024; j++)
	{
		ptrpixel[i] = ptrpixel_x[i + j*no_of_threads_interpolation + b*no_of_threads_interpolation*itration_thread];
		__syncthreads();

		//	if (i + j * n_threads< counter)
		//{
		s = ptrpixel_x + ptrpixel[i].offset;
		unsigned short Sindx = s->sample_index;
		unsigned short Lindx = s->line_index;

		//ptr = newscale;

		register float Q11 = *(newscale + (Sindx + (Lindx * 512)));
		register float Q12 = *(newscale + (Sindx + 1 + (Lindx * 512)));
		register float Q21 = *(newscale + (Sindx + ((Lindx + 1) * 512)));
		register float Q22 = *(newscale + (Sindx + 1 + ((Lindx + 1) * 512)));
		final_image[ptrpixel[i].offset] = s->c1 * Q11 + s->c2 *Q21 + s->c3 *Q12 + s->c4 *Q22;
		//final_image[i + j * 256] =150;
		//synchthreads();

		//}

	}

}


/*__global__ void interpolation_kernal(PixelDefine *ptrpixel, float *newscale, PixelDefine *s, float *ptr, BYTE *final_image)
{
//int	i = threadIdx.y + blockIdx.y *512+ blockIdx.x*512;
int	i = 0;// threadIdx.y;
//for (int j = 0; j <679; j++)
for (int j = 0; j <100024; j++)
{
if (i + j * 256< counter)
//{
s = ptrpixel + ptrpixel[i + j * 256].offset;
unsigned short Sindx = s->sample_index;
unsigned short Lindx = s->line_index;
//memset(newscale, 255, 131072 * sizeof(float));

ptr = newscale;

register float Q11 = *(ptr + (Sindx + (Lindx * 512)));
register float Q12 = *(ptr + (Sindx + 1 + (Lindx * 512)));
register float Q21 = *(ptr + (Sindx + ((Lindx + 1) * 512)));
register float Q22 = *(ptr + (Sindx + 1 + ((Lindx + 1) * 512)));
//final_image[ptrpixel[i + j * 256].offset] = s->c1 * Q11 + s->c2 *Q21 + s->c3 *Q12 + s->c4 *Q22;
final_image[j] =164;

//}
final_image[j] = 150;
}

}*/


extern "C" void
bw_filteringTest(float *data, float *filth_out, float *filtl_out)
{
	dim3 dimBlock(1, no_of_threads);
	dim3 dimGrid(1, no_of_blocks);


	bw_highpass_filterKernel << < dimGrid, dimBlock >> >(data, filth_out, filtl_out);

	//bw_lowpass_filterKernel << < dimGrid, dimBlock >> >(filth_out, filtl_out,filth_out);

}


extern "C" void
ellaptic_filteringTest(float *data, float *filth_out, float *filtl_out)
{
	dim3 dimBlock(1, no_of_threads);
	dim3 dimGrid(1, no_of_blocks);



	ellaptic_highpass_filterKernel << < dimGrid, dimBlock >> >(data, filth_out);

	ellaptic_lowpass_filterKernel << < dimGrid, dimBlock >> >(filth_out, filtl_out);

}

extern "C" void
chebychev_filteringTest(float *data, float *filth_out, float *filtl_out)
{
	dim3 dimBlock(1, no_of_threads);
	dim3 dimGrid(1, no_of_blocks);


	chebychev_highpass_filterKernel << < dimGrid, dimBlock >> >(data, filth_out);

	chebychev_lowpass_filterKernel << < dimGrid, dimBlock >> >(filth_out, filtl_out);


}
extern "C" void
least_filteringTest(float *data, float *filth_out, float *filtl_out)
{
	dim3 dimBlock(1, no_of_threads);
	dim3 dimGrid(1, no_of_blocks);


	least_highpass_filterKernel << < dimGrid, dimBlock >> >(data, filth_out);

	least_lowpass_filterKernel << < dimGrid, dimBlock >> >(filth_out, filtl_out);


}
extern "C" void
constra_filteringTest(float *data, float *filth_out, float *filtl_out)
{
	dim3 dimBlock(1, no_of_threads);
	dim3 dimGrid(1, no_of_blocks);


	constra_highpass_filterKernel << < dimGrid, dimBlock >> >(data, filth_out);

	constra_lowpass_filterKernel << < dimGrid, dimBlock >> >(filth_out, filtl_out);


}
extern "C" void
detectionTest(float *filtl_out, float *det_out, float * ph, float *z, float *lpf)
{
	dim3 dimBlock(1, no_of_threads);
	dim3 dimGrid(1, no_of_blocks);

	detection_Kernel << < dimGrid, dimBlock >> >(filtl_out, det_out, z, ph, lpf);
	//lpf_Kernel << < dimGrid, dimBlock >> >(det_out, lpf);

}

extern "C" void
avsamplingTest(float *lpf, float *sampling_out)
{
	dim3 dimBlock(1, no_of_threads);
	dim3 dimGrid(1, no_of_blocks);


	av_sampling_Kernel << < dimGrid, dimBlock >> > (lpf, sampling_out);


}

extern "C" void
peaksamplingTest(float *lpf, float *sampling_out)
{
	dim3 dimBlock(1, no_of_threads);
	dim3 dimGrid(1, no_of_blocks);

	peak_sampling_Kernel << < dimGrid, dimBlock >> > (lpf, sampling_out);


}


extern "C" void
samplingTest(float *lpf, float *sampling_out)
{
	dim3 dimBlock(1, no_of_threads);
	dim3 dimGrid(1, no_of_blocks);


	sampling_Kernel << < dimGrid, dimBlock >> > (lpf, sampling_out);

}



extern "C" void
dynamicTest(float *new_scale, float *sampling_out, int *dB_v, float *dynamic, float *max)
{
	dim3 dimBlock(1, no_of_threads);
	dim3 dimGrid(1, no_of_blocks);

	dynamic_Kernel << < dimGrid, dimBlock >> > (new_scale, sampling_out, dB_v, dynamic, max);

}


extern "C" void
interpolationTest(PixelDefine *pixel, float *new_scale, PixelDefine *s, BYTE *FI)
{
	dim3 dimBlock(1, no_of_threads_interpolation);
	dim3 dimGrid(1, no_of_blocks_interpolation);


	interpolation_kernal << < dimGrid, dimBlock >> > (pixel, new_scale, s, FI);

	//float *newscale = 0;
	//hipMalloc(&newscale, 131072 * sizeof(float));


}

/*extern "C" void
interpolationTest(PixelDefine *ptrpixel, BYTE *FINAL)
{
dim3 dimBlock(1, 1);
dim3 dimGrid(1, no_of_blocks);

float*new_scale = 0;
PixelDefine *s = 0;
float *ptr = 0;
//float * NS = 0;
BYTE *FI = 0;
PixelDefine *pixel = 0;

hipMalloc(&pixel, TOTAL_PIXEL * sizeof(PixelDefine));
hipMalloc(&FI, TOTAL_PIXEL * CHANNELCOUNT * sizeof(BYTE));
hipMalloc(&ptr, IMSIZE * sizeof(float));
hipMalloc(&s, TOTAL_PIXEL * sizeof(PixelDefine));
//hipMemcpy(pixel, ptrpixel, 512 * 512 * sizeof(PixelDefine), hipMemcpyHostToDevice);
hipMalloc(&new_scale, IMSIZE * sizeof(float));


interpolation_kernal <<< dimGrid, dimBlock >>> (pixel, new_scale, s, ptr, FI);
//float *newscale = 0;
//hipMalloc(&newscale, 131072 * sizeof(float));
//memset(new_scale, 255, 131072 * sizeof(float));
//hipMemcpy(FINAL , FI, 512 * 512 * 3 * sizeof(BYTE), hipMemcpyDeviceToHost);
bool error = false;

//hipMemcpy( FI, FINAL, 512 * 512 * 3 * 1, hipMemcpyHostToDevice);
FINAL[0] = 5;
FINAL[1]=7;

hipMemcpy(FINAL, FI, 512 * 512 * 3 * 1, hipMemcpyDeviceToHost);
int k = 0;


}*/

extern "C" float
interpolation_performance(PixelDefine *pixel, float *new_scale, PixelDefine *s, BYTE *FI)
{
	dim3 dimBlock(1, no_of_threads_interpolation);
	dim3 dimGrid(1, no_of_blocks_interpolation);


	hipDeviceSynchronize();
	hipError_t error;
	// Allocate CUDA events that we'll use for timing
	hipEvent_t start;
	error = hipEventCreate(&start);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	hipEvent_t stop;
	error = hipEventCreate(&stop);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Record the start event
	error = hipEventRecord(start, NULL);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Execute the kernel
	int nIter = 300;

	for (int j = 0; j < nIter; j++)
	{
		interpolation_kernal << < dimGrid, dimBlock >> > (pixel, new_scale, s, FI);
	}

	// Record the stop event
	error = hipEventRecord(stop, NULL);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Wait for the stop event to complete
	error = hipEventSynchronize(stop);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	float msecTotal = 0.0f;
	error = hipEventElapsedTime(&msecTotal, start, stop);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Compute and print the performance
	//double flopsPerMatrixMul = 2.0 * (double)dimsA.x * (double)dimsA.y * (double)dimsB.x;
	float msecinterpolation = msecTotal / nIter;
	//	double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);

	//printf(" Time= %.3f msec, ", msecPerMatrixMul);

	return msecinterpolation;
}

extern "C" float
dynamic_performance(float *new_scale, float *sampling_out, int *dB_v, float *dynamic, float *max)
{
	dim3 dimBlock(1, no_of_threads);
	dim3 dimGrid(1, no_of_blocks);


	hipDeviceSynchronize();
	hipError_t error;
	// Allocate CUDA events that we'll use for timing
	hipEvent_t start;
	error = hipEventCreate(&start);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	hipEvent_t stop;
	error = hipEventCreate(&stop);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Record the start event
	error = hipEventRecord(start, NULL);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Execute the kernel
	int nIter = 100;

	for (int j = 0; j < nIter; j++)
	{

		dynamic_Kernel << < dimGrid, dimBlock >> > (new_scale, sampling_out, dB_v, dynamic, max);

	}

	// Record the stop event
	error = hipEventRecord(stop, NULL);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Wait for the stop event to complete
	error = hipEventSynchronize(stop);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	float msecTotal = 0.0f;
	error = hipEventElapsedTime(&msecTotal, start, stop);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Compute and print the performance
	//double flopsPerMatrixMul = 2.0 * (double)dimsA.x * (double)dimsA.y * (double)dimsB.x;
	float msecinterpolation = msecTotal / nIter;
	//	double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);

	//printf(" Time= %.3f msec, ", msecPerMatrixMul);

	return msecinterpolation;
}

extern "C" float
undersampling_performance(float *lpf, float *sampling_out)
{
	dim3 dimBlock(1, no_of_threads);
	dim3 dimGrid(1, no_of_blocks);


	hipDeviceSynchronize();
	hipError_t error;
	// Allocate CUDA events that we'll use for timing
	hipEvent_t start;
	error = hipEventCreate(&start);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	hipEvent_t stop;
	error = hipEventCreate(&stop);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Record the start event
	error = hipEventRecord(start, NULL);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Execute the kernel
	int nIter = 100;

	for (int j = 0; j < nIter; j++)
	{

		peak_sampling_Kernel << < dimGrid, dimBlock >> > (lpf, sampling_out);

	}

	// Record the stop event
	error = hipEventRecord(stop, NULL);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Wait for the stop event to complete
	error = hipEventSynchronize(stop);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	float msecTotal = 0.0f;
	error = hipEventElapsedTime(&msecTotal, start, stop);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Compute and print the performance
	//double flopsPerMatrixMul = 2.0 * (double)dimsA.x * (double)dimsA.y * (double)dimsB.x;
	float msecinterpolation = msecTotal / nIter;
	//	double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);

	//printf(" Time= %.3f msec, ", msecPerMatrixMul);

	return msecinterpolation;
}

extern "C" float
detection_performance(float *filtl_out, float *det_out, float * ph, float *z, float *lpf)
{
	dim3 dimBlock(1, no_of_threads);
	dim3 dimGrid(1, no_of_blocks);


	hipDeviceSynchronize();
	hipError_t error;
	// Allocate CUDA events that we'll use for timing
	hipEvent_t start;
	error = hipEventCreate(&start);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	hipEvent_t stop;
	error = hipEventCreate(&stop);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Record the start event
	error = hipEventRecord(start, NULL);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Execute the kernel
	int nIter = 100;

	for (int j = 0; j < nIter; j++)
	{
		detection_Kernel << < dimGrid, dimBlock >> >(filtl_out, det_out, z, ph, lpf);


	}

	// Record the stop event
	error = hipEventRecord(stop, NULL);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Wait for the stop event to complete
	error = hipEventSynchronize(stop);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	float msecTotal = 0.0f;
	error = hipEventElapsedTime(&msecTotal, start, stop);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Compute and print the performance
	//double flopsPerMatrixMul = 2.0 * (double)dimsA.x * (double)dimsA.y * (double)dimsB.x;
	float msecinterpolation = msecTotal / nIter;
	//	double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);

	//printf(" Time= %.3f msec, ", msecPerMatrixMul);

	return msecinterpolation;
}

extern "C" float
filtering_performance(float *data, float *filth_out, float *filtl_out)
{
	dim3 dimBlock(1, no_of_threads);
	dim3 dimGrid(1, no_of_blocks);


	hipDeviceSynchronize();
	hipError_t error;
	// Allocate CUDA events that we'll use for timing
	hipEvent_t start;
	error = hipEventCreate(&start);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	hipEvent_t stop;
	error = hipEventCreate(&stop);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Record the start event
	error = hipEventRecord(start, NULL);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Execute the kernel
	int nIter = 100;

	for (int j = 0; j < nIter; j++)
	{

		bw_highpass_filterKernel << < dimGrid, dimBlock >> >(data, filth_out, filtl_out);

	}

	// Record the stop event
	error = hipEventRecord(stop, NULL);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Wait for the stop event to complete
	error = hipEventSynchronize(stop);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	float msecTotal = 0.0f;
	error = hipEventElapsedTime(&msecTotal, start, stop);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Compute and print the performance
	//double flopsPerMatrixMul = 2.0 * (double)dimsA.x * (double)dimsA.y * (double)dimsB.x;
	float msecinterpolation = msecTotal / nIter;
	//	double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);

	//printf(" Time= %.3f msec, ", msecPerMatrixMul);

	return msecinterpolation;
}